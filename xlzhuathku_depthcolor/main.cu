
#include <hip/hip_runtime.h>
#include<iostream>

__global__ void kernel(void) {
}

int main()
{
	kernel<<<1,1>>>();
	printf("Hello world!\n");

	int count;
	hipGetDeviceCount(&count);
	printf("Count:\t%d\n", count);

	hipDeviceProp_t prop;
	for( int i=0; i<count; i++) {
		hipGetDeviceProperties(&prop,i);
		//print info
		printf("Name:\t%s\n", prop.name);
		printf("Global Mem:\t%d\n", prop.totalGlobalMem);
		printf("Const Mem:\t%d\n", prop.totalConstMem);
		printf("Shared Mem:\t%d\n", prop.sharedMemPerBlock);
		printf("Mem Pitch:\t%d\n", prop.memPitch);
		printf("Max Threads:\t%d\n", prop.maxThreadsPerBlock);
		printf("Compute Capasity:\t%d,%d\n", prop.minor, prop.major);
		printf("Processor Num:\t%d,%d\n", prop.multiProcessorCount);
	}
	return 0;
}