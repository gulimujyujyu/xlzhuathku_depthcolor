#include "hip/hip_runtime.h"
#include "randomforest_id3.h"
#include <stdio.h>

__global__ void debugger( float *ptr) {
	int tid = blockIdx.x;
	if( tid < 25)
		ptr[tid] += 1;
}

void readTrainingSet(float **a, char *filename, int itemNum, int dim)
{
	FILE *file;
	file = fopen( filename, "r");

	*a = (float *)malloc( itemNum*(dim+1)*sizeof(float));

	for ( int i=0; i<itemNum; i++) {
		for ( int j=0; j<dim; j++) {
			fscanf( file, "%f", &((*a)[i*(dim+1)+j]));
			//printf( "%.2f ", (*a)[i*(dim+1)+j]);
		}
		fscanf( file, "%f", &((*a)[i*(dim+1)+dim]));
		//printf( "%.2f\n", (*a)[i*(dim+1)+dim]);
	}
	fclose(file);
}

float * createDevicePtr(float *a, int itemNum, bool needRandom, int dim)
{
	// Host code
	float *devPtr;
	float *b;
	b = (float *)malloc( itemNum*(dim+1)*sizeof(float));
	int width = (dim+1), height = itemNum;
	
	size_t pitch;
	hipMalloc(&devPtr, width * height * sizeof(float));
	hipMemcpy(devPtr, a,  width * height * sizeof(float), hipMemcpyHostToDevice);
    
	for ( int i=0; i<itemNum; i++) {
		for ( int j=0; j<dim; j++) {
			printf( "%f ", a[i*(dim+1)+j]);
		}
		printf( "%f\n", a[i*(dim+1)+dim]);
	}
	//debugger<<<25,1>>>(devPtr);
	//hipMemcpy(b, devPtr,  width * height * sizeof(float), hipMemcpyDeviceToHost);
	//for ( int i=0; i<itemNum; i++) {
	//	for ( int j=0; j<dim; j++) {
	//		printf( "%f ", b[i*(dim+1)+j]);
	//	}
	//	printf( "%f\n", b[i*(dim+1)+dim]);
	//}
	return devPtr;
}