#include "hip/hip_runtime.h"
#include<iostream>
#include "randomforest_id3.h"
#include "id3tree.h"
#include <stdio.h>

__global__ void kernel(void) {
}

int main()
{
	float *tptr;
	float *devPtr;
	char filename[] = "test.txt";
	int dim = 9;
	int itemNum = 5;
	readTrainingSet( &tptr, filename, itemNum, dim);
	//
	//for ( int i=0; i<itemNum; i++) {
	//	for ( int j=0; j<dim; j++) {
	//		printf( "%f ", tptr[i*(dim+1)+j]);
	//	}
	//	printf( "%f\n", tptr[i*(dim+1)+dim]);
	//}
	devPtr = createDevicePtr( tptr, itemNum, false, dim);

	Forest forest;
	trainForest( &forest, devPtr,itemNum, dim);

	return 0;
}

/*
int main()
{
	//STEP1: read images
	
	//STEP2: train random forest classifier

	//STEP3: test classifier
	kernel<<<1,1>>>();
	printf("Hello world!\n");

	int count;
	hipGetDeviceCount(&count);
	printf("Count:\t%d\n", count);

	hipDeviceProp_t prop;
	for( int i=0; i<count; i++) {
		hipGetDeviceProperties(&prop,i);
		//print info
		printf("Name:\t%s\n", prop.name);
		printf("Global Mem:\t%d\n", prop.totalGlobalMem);
		printf("Const Mem:\t%d\n", prop.totalConstMem);
		printf("Shared Mem:\t%d\n", prop.sharedMemPerBlock);
		printf("Mem Pitch:\t%d\n", prop.memPitch);
		printf("Max Threads:\t%d\n", prop.maxThreadsPerBlock);
		printf("Compute Capasity:\t%d,%d\n", prop.minor, prop.major);
		printf("Processor Num:\t%d,%d\n", prop.multiProcessorCount);
	}
	return 0;
}
*/