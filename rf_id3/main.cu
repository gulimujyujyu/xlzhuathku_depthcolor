#include "hip/hip_runtime.h"
#include<iostream>
#include "id3tree_gpu.h"
#include <stdio.h>

__global__ void kernel(void) {
}

int main()
{
	Forest frst;
	char filename[] = "data/000000.tree";

	printf("%s\n",filename);
	readForest(filename, &frst);
	printForest(&frst);

	return 0;
}

/*
int main()
{
	//STEP1: read images
	
	//STEP2: train random forest classifier

	//STEP3: test classifier
	kernel<<<1,1>>>();
	printf("Hello world!\n");

	int count;
	hipGetDeviceCount(&count);
	printf("Count:\t%d\n", count);

	hipDeviceProp_t prop;
	for( int i=0; i<count; i++) {
		hipGetDeviceProperties(&prop,i);
		//print info
		printf("Name:\t%s\n", prop.name);
		printf("Global Mem:\t%d\n", prop.totalGlobalMem);
		printf("Const Mem:\t%d\n", prop.totalConstMem);
		printf("Shared Mem:\t%d\n", prop.sharedMemPerBlock);
		printf("Mem Pitch:\t%d\n", prop.memPitch);
		printf("Max Threads:\t%d\n", prop.maxThreadsPerBlock);
		printf("Compute Capasity:\t%d,%d\n", prop.minor, prop.major);
		printf("Processor Num:\t%d,%d\n", prop.multiProcessorCount);
	}
	return 0;
}
*/