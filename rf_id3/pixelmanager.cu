#include "hip/hip_runtime.h"
#include "pixelmanager.h"
#include "id3tree_gpu.h"



void createGPUForest( Forest *forest, float **a)
{
	int treeNum;
	int maxTreeNodeNum;
	int treeNodeSize;
	int i;
	int j;
	int k;

	treeNum = NUM_OF_TREES;
	maxTreeNodeNum = MAX_TREE_NODE_NUM;
	for( i=0; i<treeNum; i++) {
		if( maxTreeNodeNum < forest->forest[i].numOfNodes)
			maxTreeNodeNum = forest->forest[i].numOfNodes;
	}
	treeNodeSize = 5 + NUM_OF_CLASSES;

	*a = (float *)malloc(NUM_OF_TREES * maxTreeNodeNum * treeNodeSize * sizeof(float));

	for( i=0; i<treeNum; i++) {
		for( j=0; j<forest->forest[i].numOfNodes; j++) {
			(*a)[ i*(maxTreeNodeNum*treeNodeSize) + j*treeNodeSize + 0] = forest->forest[i].nodeArray[j].parent;
			(*a)[ i*(maxTreeNodeNum*treeNodeSize) + j*treeNodeSize + 1] = forest->forest[i].nodeArray[j].leftChild;
			(*a)[ i*(maxTreeNodeNum*treeNodeSize) + j*treeNodeSize + 2] = forest->forest[i].nodeArray[j].isLeaf;
			(*a)[ i*(maxTreeNodeNum*treeNodeSize) + j*treeNodeSize + 3] = forest->forest[i].nodeArray[j].attri;
			(*a)[ i*(maxTreeNodeNum*treeNodeSize) + j*treeNodeSize + 4] = forest->forest[i].nodeArray[j].thres;
			for( k=0; k<NUM_OF_CLASSES;k++) {
				(*a)[ i*(maxTreeNodeNum*treeNodeSize) + j*treeNodeSize + 4+k] = forest->forest[i].nodeArray[j].distribution[k];
			}
		}
	}
	/*
	printf( "%d ", a->parent);
	printf( "%d ", a->leftChild);
	printf( "%d ", a->isLeaf);
	printf( "%d ", a->attri);
	printf( "%f ", a->thres);
	for( i=0; i<NUM_OF_CLASSES;i++) {
	printf("%f ", a->distribution[i]);
	}
	printf( "\n");
	*/
}

void readParamToGPU( char *filename, int **a)
{
	FILE *pFile = NULL;
	int i;

	pFile = fopen(filename, "r");

	if(!pFile) return;
	*a = (int *)malloc(2*NUM_OF_PARAMS*sizeof(int));

	for( i=0; i<NUM_OF_PARAMS; i++) {
		fscanf(pFile, "%d %d", &((*a)[i*2]), &((*a)[i*2+1]));
	}
	fclose(pFile);
}

void createGPUImage( char *filename, float **a)
{
	FILE *pFile = NULL;
	int i;

	pFile = fopen(filename, "r");

	if(!pFile) return;
	*a = (float *)malloc(512*512*sizeof(float));

	for( i=0; i<512*512; i++) {
		fscanf(pFile, "%f", &((*a)[i]));
	}
	fclose(pFile);
}

__global__ void predictGPU( float* img, int* img_param, float * forest)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	float pixel = img[x*512+y];
	int iLevel;
	int ii;
	int iTree;
	bool run = true;
	float maxValue = -1;
	int maxLabel = -1;
	int blk = MAX_TREE_NODE_NUM*(5+NUM_OF_CLASSES);
	int lin = (5+NUM_OF_CLASSES);
	float dis[NUM_OF_CLASSES] = {0};

	run = pixel < 0.5? false:true;
	pixel = 255-pixel;

	/*
	p1 = (float)(((uchar*)(pImg->imageData + pImg->widthStep*yy))[xx*3]);
	if ( p1 < 0.5)
	continue;
	p1 = 255-p1;
	for ( int i=0; i<SEEDS_ARRAY_SIZE; i++) {
	//check
	u1 = xx + seedsX[i]/p1;
	if ( u1 < 0 || u1 >= width) 
	p2 = MAX_DEPTHVALUE;
	else 
	p2 = (float)(((uchar*)(pImg->imageData + pImg->widthStep*yy))[u1*3]);

	v1 = yy + seedsY[i]/p1;
	if ( v1 < 0 || v1 >= height)
	p3 = MAX_DEPTHVALUE;
	else
	p3 = (float)(((uchar*)(pImg->imageData + pImg->widthStep*yy))[v1*3]);

	p2 = p2<0.5? MAX_DEPTHVALUE: (255-p2);
	p3 = p3<0.5? MAX_DEPTHVALUE: (255-p3);
	feat = p2 - p3;
	outfile << feat << "\t";				
	}
	*/

	for(iTree = 0; run && iTree < NUM_OF_TREES; iTree++) {
		for( iLevel = 0; run;) {
			iLevel = forest[iTree*blk + iLevel*lin + 1];
			run = (forest[iTree*blk + iLevel*lin + 2] != 0);
		}
		run = true;
		for( ii = 0; ii < NUM_OF_CLASSES; ii ++) {
			dis[ii] += forest[iTree*blk + iLevel*lin + 4+ii];
		}
	}

	for( ii = 0; ii < NUM_OF_CLASSES; ii ++) 
		maxLabel = dis[ii] > maxValue? ii: maxLabel;

}

void predictCPU( float* img, int* img_param, float * forest)
{
	int xx;
	int yy;
	float pixel;
	int iLevel;
	int ii;
	int iTree;
	bool run = true;
	float maxValue = -1;
	int maxLabel = -1;
	int blk = MAX_TREE_NODE_NUM*(5+NUM_OF_CLASSES);
	int lin = (5+NUM_OF_CLASSES);
	float dis[NUM_OF_CLASSES] = {0};

	for(xx=0; xx<512; xx++) 
		for(yy=0; yy<512; yy++){
			pixel = img[xx*512+yy];
			run = true;
			maxValue = -1;
			maxLabel = -1;
			for( ii = 0; ii < NUM_OF_CLASSES; ii ++) 
				dis[ii] = 0;
			if(pixel < 0.5)
				run = false;
			pixel = 255-pixel;

			for(iTree = 0; run && iTree < NUM_OF_TREES; iTree++) {
				for( iLevel = 0; run;) {
					iLevel = forest[iTree*blk + iLevel*lin + 1];
					run = (forest[iTree*blk + iLevel*lin + 2] != 0);
				}
				run = true;
				for( ii = 0; ii < NUM_OF_CLASSES; ii ++) {
					dis[ii] += forest[iTree*blk + iLevel*lin + 4+ii];
				}
			}

			for( ii = 0; ii < NUM_OF_CLASSES; ii ++) 
				if(dis[ii] > maxValue) {
					maxLabel = ii;
				}
		}
}
